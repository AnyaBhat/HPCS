
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4  // Number of columns

__global__ void addRow(float *A, float *B, float *C, int rows) {
    int row = blockIdx.x;  // Each thread handles one row
    if (row < rows) {
        for (int col = 0; col < N; col++) {
            C[row * N + col] = A[row * N + col] + B[row * N + col];
        }
    }
}

int main() {
    int rows = 4;
    float h_A[rows][N], h_B[rows][N], h_C[rows][N];
    float *d_A, *d_B, *d_C;


    
    printf("\nMatrix A:\n");
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < N; ++j) {
            h_A[i][j] = (float)(i + j);
            printf("%.2f ",h_A[i][j]);
        }
        printf("\n");
    }
    printf("\nMatrix B:\n");
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < N; ++j) {
            h_B[i][j] = (float)(i * j);
            printf("%.2f ",h_B[i][j]);
        }
        printf("\n");
    }
    
    hipMalloc((void**)&d_A, rows * N * sizeof(float));
    hipMalloc((void**)&d_B, rows * N * sizeof(float));
    hipMalloc((void**)&d_C, rows * N * sizeof(float));

    hipMemcpy(d_A, h_A, rows * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, rows * N * sizeof(float), hipMemcpyHostToDevice);

    addRow<<<rows, 1>>>(d_A, d_B, d_C, rows);

    hipMemcpy(h_C, d_C, rows * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    printf("Resultant Matrix C (Row-wise Addition):\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_C[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}

