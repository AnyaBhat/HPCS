#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 5 // Size of the matrix (N x N)

__global__ void matrixAdd(float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        C[row * N + col] = A[row * N + col] + B[row * N + col];
    }
}

__global__ void matrixMul(float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float value = 0;
    if (row < N && col < N) {
        for (int k = 0; k < N; k++) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

void initializeMatrix(float *matrix, int size) {
    for (int i = 0; i < size; ++i) {
        matrix[i] = (float)(rand() % 10);
    }
}

void printMatrix(float *matrix) {
    for (int i = 0; i < N * N; ++i) {
        printf("%f ", matrix[i]);
        if ((i + 1) % N == 0) printf("\n");
    }
}

int main() {
    // Allocate host memory
    size_t size = N * N * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C_add = (float *)malloc(size);
    float *h_C_mul = (float *)malloc(size);

    // Initialize matrices
    initializeMatrix(h_A, N * N);
    initializeMatrix(h_B, N * N);

    // Allocate device memory
    float *d_A, *d_B, *d_C_add, *d_C_mul;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C_add, size);
    hipMalloc((void**)&d_C_mul, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 threadsPerBlock(4, 4);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch matrix addition kernel
    matrixAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C_add);
    hipDeviceSynchronize();

    // Launch matrix multiplication kernel
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C_mul);
    hipDeviceSynchronize();

    // Copy results back to host
    hipMemcpy(h_C_add, d_C_add, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_C_mul, d_C_mul, size, hipMemcpyDeviceToHost);

    // Print results
    printf("Matrix A:\n");
    printMatrix(h_A);
    printf("Matrix B:\n");
    printMatrix(h_B);
    printf("Matrix Addition Result:\n");
    printMatrix(h_C_add);
    printf("Matrix Multiplication Result:\n");
    printMatrix(h_C_mul);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C_add);
    free(h_C_mul);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C_add);
    hipFree(d_C_mul);

    return 0;
}

/*
Matrix A:
3.000000 6.000000 7.000000 5.000000 3.000000 
5.000000 6.000000 2.000000 9.000000 1.000000 
2.000000 7.000000 0.000000 9.000000 3.000000 
6.000000 0.000000 6.000000 2.000000 6.000000 
1.000000 8.000000 7.000000 9.000000 2.000000 
Matrix B:
0.000000 2.000000 3.000000 7.000000 5.000000 
9.000000 2.000000 2.000000 8.000000 9.000000 
7.000000 3.000000 6.000000 1.000000 2.000000 
9.000000 3.000000 1.000000 9.000000 4.000000 
7.000000 8.000000 4.000000 5.000000 0.000000 
Matrix Addition Result:
3.000000 8.000000 10.000000 12.000000 8.000000 
14.000000 8.000000 4.000000 17.000000 10.000000 
9.000000 10.000000 6.000000 10.000000 5.000000 
15.000000 3.000000 7.000000 11.000000 10.000000 
8.000000 16.000000 11.000000 14.000000 2.000000 
Matrix Multiplication Result:
169.000000 78.000000 80.000000 136.000000 103.000000 
156.000000 63.000000 52.000000 171.000000 119.000000 
165.000000 69.000000 41.000000 166.000000 109.000000 
102.000000 84.000000 80.000000 96.000000 50.000000 
216.000000 82.000000 78.000000 169.000000 127.000000 


*/
