
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4  // Number of rows

__global__ void addColumn(float *A, float *B, float *C, int cols) {
    int col = blockIdx.x;  // Each thread handles one column
    if (col < cols) {
        for (int row = 0; row < N; row++) {
            C[row * cols + col] = A[row * cols + col] + B[row * cols + col];
        }
    }
}

int main() {
    int cols = 4;
    float h_A[N][cols], h_B[N][cols], h_C[N][cols];
    float *d_A, *d_B, *d_C;

    // Initialize matrices A and B
    printf("\nMatrix A:\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < cols; ++j) {
            h_A[i][j] = (float)(i + j);
            printf("%.2f ",h_A[i][j]);
        }
        printf("\n");
    }
    printf("\nMatrix B:\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < cols; ++j) {
            h_B[i][j] = (float)(i * j);
            printf("%.2f ",h_B[i][j]);
        }
        printf("\n");
    }

    hipMalloc((void**)&d_A, N * cols * sizeof(float));
    hipMalloc((void**)&d_B, N * cols * sizeof(float));
    hipMalloc((void**)&d_C, N * cols * sizeof(float));

    hipMemcpy(d_A, h_A, N * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * cols * sizeof(float), hipMemcpyHostToDevice);

    addColumn<<<cols, 1>>>(d_A, d_B, d_C, cols);

    hipMemcpy(h_C, d_C, N * cols * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    printf("Resultant Matrix C (Column-wise Addition):\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", h_C[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
/*
Matrix A:
0.00 1.00 2.00 3.00 
1.00 2.00 3.00 4.00 
2.00 3.00 4.00 5.00 
3.00 4.00 5.00 6.00 

Matrix B:
0.00 0.00 0.00 0.00 
0.00 1.00 2.00 3.00 
0.00 2.00 4.00 6.00 
0.00 3.00 6.00 9.00 
Resultant Matrix C (Column-wise Addition):
0.000000 1.000000 2.000000 3.000000 
1.000000 3.000000 5.000000 7.000000 
2.000000 5.000000 8.000000 11.000000 
3.000000 7.000000 11.000000 15.000000 

*/
