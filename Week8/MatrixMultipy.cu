#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void matrixMulKernel(float *A, float *B, float *C, int M, int N, int P) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < P) {
        float value = 0;
        for (int i = 0; i < N; ++i) {
            value += A[row * N + i] * B[i * P + col];
        }
        C[row * P + col] = value;
    }
}
int main() {
    int M, N, P;

    printf("Enter the number of rows in matrix A (M): ");
    scanf("%d", &M);
    printf("Enter the number of columns in matrix A / rows in matrix B (N): ");
    scanf("%d", &N);
    printf("Enter the number of columns in matrix B (P): ");
    scanf("%d", &P);

    float *h_A = (float*) malloc(M * N * sizeof(float));
    float *h_B = (float*) malloc(N * P * sizeof(float));
    float *h_C = (float*) malloc(M * P * sizeof(float));

    printf("Enter elements of matrix A (%d x %d):\n", M, N);
    for (int i = 0; i < M * N; ++i) {
        scanf("%f", &h_A[i]);
    }

    printf("Enter elements of matrix B (%d x %d):\n", N, P);
    for (int i = 0; i < N * P; ++i) {
        scanf("%f", &h_B[i]);
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M * N * sizeof(float));
    hipMalloc((void**)&d_B, N * P * sizeof(float));
    hipMalloc((void**)&d_C, M * P * sizeof(float));

    hipMemcpy(d_A, h_A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * P * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((P + dimBlock.x - 1) / dimBlock.x, (M + dimBlock.y - 1) / dimBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    clock_t totalStart = clock();
    hipEventRecord(start, 0);

    matrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, N, P);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);

    hipMemcpy(h_C, d_C, M * P * sizeof(float), hipMemcpyDeviceToHost);

    clock_t totalEnd = clock();
    double totalTime = ((double)(totalEnd - totalStart)) / CLOCKS_PER_SEC;

    printf("Resultant matrix C (%d x %d):\n", M, P);
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < P; ++j) {
            printf("%f ", h_C[i * P + j]);
        }
        printf("\n");
    }

    printf("Kernel execution time: %f ms\n", kernelTime);
    printf("Total execution time: %f seconds\n", totalTime);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
