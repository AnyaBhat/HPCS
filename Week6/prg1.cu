#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addarr( float *a, float *b, float *c,int N){
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	if(tid<N){
	 	c[tid]=a[tid]+b[tid];
	}
}

int main(){
	int N = 5;  // Size of arrays
    	int size = N * sizeof(float);

    	float h_a[N]={1.0,2.0,3.0,4.0,5.0};
    	float h_b[N]={6.0,7.0,8.0,9.0,10.0};
    	float h_c[N];

    	float *d_a, *d_b, *d_c;
	
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);
	
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
	
	addarr<<<N,1>>>(d_a,d_b,d_c,N);
	
	hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);
	
	for(int i=0;i<N;i++){
		printf("%f + %f = %f \n",h_a[i],h_b[i],h_c[i]);
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
