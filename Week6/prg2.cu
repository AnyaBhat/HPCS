#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void deci_oct ( int *ip, int *op,int len){
	int tid=threadIdx.x;
	if(tid <len){
		int decNum = ip[tid];
        	int octNum = 0;
        	int factor = 1;

        	while (decNum > 0) {
            		int remainder = decNum % 8;
            		octNum += remainder * factor;
            		factor *= 10;
            		decNum /= 8;
        	}
        	op[tid] = octNum;
	}
}

int main(){
	int N = 5;  // Size of arrays
    	int size = N * sizeof(int);

    	int ip[N]={9,8,33,40,54};
    	int op[N];

    	int *d_ip, *d_op;
	
	hipMalloc((void **)&d_ip,size);
	hipMalloc((void **)&d_op,size);
	
	hipMemcpy(d_ip, ip, size, hipMemcpyHostToDevice);
	
	
	deci_oct<<<1,N>>>(d_ip,d_op,N);
	
	hipMemcpy(op,d_op,size,hipMemcpyDeviceToHost);
	
	for(int i=0;i<N;i++){
		printf("octal(%d) = %d \n",ip[i],op[i]);
	}
	hipFree(d_ip);
	hipFree(d_op);
	return 0;
}
