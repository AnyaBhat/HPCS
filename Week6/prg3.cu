#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void swap ( int *ip, int len){
	int tid=threadIdx.x*2;
	if(tid <len-1){
		int temp=ip[tid];
		ip[tid]=ip[tid+1];
		ip[tid+1]=temp;
	}
}

int main(){
	int N = 6;  // Size of arrays
    	int size = N * sizeof(int);

    	int ip[N]={10,5,20,10,15,7};

    	int *d_ip;
	
	hipMalloc((void **)&d_ip,size);
	
	hipMemcpy(d_ip, ip, size, hipMemcpyHostToDevice);
	printf("Orginal array: ");
	for(int i=0;i<N;i++){
		printf("%d ",ip[i]);
	}
	printf("\n");
	int n=N/2;
	swap<<<1,n>>>(d_ip,N);
	
	hipMemcpy(ip,d_ip,size,hipMemcpyDeviceToHost);
	printf("Swapped array:");
	for(int i=0;i<N;i++){
		printf("%d ",ip[i]);
	}
	printf("\n");
	hipFree(d_ip);
	return 0;
}
